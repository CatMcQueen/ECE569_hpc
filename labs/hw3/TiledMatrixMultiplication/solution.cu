#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use tiling with shared memory for arbitrary size
  //int TILE_WIDTH = numCCols/blockDim.x;
  int width      = numBRows;

  // create the tiles in shared memory
  __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

  // read in the variables 
  int tx = threadIdx.x;
  int ty = threadIdx.y; 
  int bx = blockIndex.x;
  int by = blockIndex.y;  

  // get the row and column indexes to process  
  int row    = by * TILE_WIDTH + ty;
  int col    = bx * TILE_WIDTH + tx;
  float cval = 0;  

  // for each tile in the images
  for (int p = 0; p < width/TILE_WIDTH; p++) {

    // read in the right values for that tile and put them in the correct
    // location in the shared memory
    ds_A[ty][tx] = A[row*width + p*TILE_WIDTH + tx]
    ds_B[ty][tx] = B[(p*TILE_WIDTH +ty) * width + col]  
      
    // then process on the tile of data from shared memory for the partial product
    for (int i=0; i < TILE_WIDTH; i++) {
      cval += ds_A[ty][i] * ds_B[i][tx];
    }

    // set the correct memory location
    C[row*width*col] = cval;
  }
  
  
  
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C(you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
                            
  //@@ Set numCRows and numCColumns
  numCRows    = 0;   // set to correct value
  numCColumns = 0;   // set to correct value
  //@@ Allocate the hostC matrix
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceA, numARows*numAColums*sizeof(float));
  hipMalloc((void **) &deviceB, numBRows*numBColums*sizeof(float));
  hipMalloc((void **) &deviceC, numARows*numBColums*sizeof(float));


  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numAColumns*numARows*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBColumns*numBRows*sizeof(float), hipMemcpyHostToDevice);  
  

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // note that TILE_WIDTH is set to 16 on line number 13. 
  dim3 BlockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 GridDim;
  
  // rows = y, column = x
  // do it in terms of C (outside edges of matrix multiply)
  // then add one to round up? maybe add .5?
  GridDim.x = (numCColumns - 1) / BlockDim.x + 1;
  GridDim.y = (numCRows - 1) / BlockDim.y + 1;
  
  
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<GridDim, BlockDim>>>(deviceA, deviceB, deviceC, 
				numARows, numAColumns, 
				numBRows, numBColumns,
				numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCColumns*numCRows*sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  wbTime_stop(GPU, "Freeing GPU Memory");
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
