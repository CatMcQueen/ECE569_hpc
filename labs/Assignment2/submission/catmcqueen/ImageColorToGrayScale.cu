#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


//@@ INSERT DEVICE CODE HERE

__global__ void ColorToGrayscale(float *inImg, float *outImg, int width, int height) {
        int idx, grayidx;
        int col = blockDim.x * blockIdx.x + threadIdx.x; // row of image
        int row  = blockDim.y * blockIdx.y + threadIdx.y; // col of image
        int numchannel = 3; // since it's RGB there are 3 channels

        // x = col and y = row
        if (col < width && row < height) {
                // each spot is 3 big (rgb) so get the number of spots
                grayidx = row * width + col;
                idx     = grayidx * numchannel; // and multiply by three to get current index
                // to calculate the beginning of the 3 for that pixel
                float r = inImg[idx];           //red
                float g = inImg[idx + 1];       //green
                float b = inImg[idx + 2];       //blue
                outImg[grayidx]  = (0.21*r + 0.71*g + 0.07*b); // now convert to grayscale
        }
}


int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE

  // 256 = 16 * 16
  dim3 BlockDim(16,16);
  dim3 GridDim; // gridDim set belo

  // set grid to work in x & y to run rows and columns both
  GridDim.x = (imageWidth + BlockDim.x - 1) / BlockDim.x;
  GridDim.y = (imageHeight + BlockDim.y - 1) / BlockDim.y;

  // call the greyscale function
  ColorToGrayscale<<<GridDim, BlockDim>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
